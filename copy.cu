#include "hip/hip_runtime.h"


int copyToDevice()
{

	hipMemcpyToSymbol(HIP_SYMBOL(dl1),h_dl1, sizeof(float)*8*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(dl2),h_dl2, sizeof(float)*8*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(dl3),h_dl3, sizeof(float)*8*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(dl4),h_dl4, sizeof(float)*8*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(dl5),h_dl5, sizeof(float)*8*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(dl6),h_dl6, sizeof(float)*8*MXMED);

	//hipMemcpyToSymbol(HIP_SYMBOL(alphi),h_alphi , sizeof(float)*2*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(bpar), h_bpar, sizeof(float)*2*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(delpos), h_delpos, sizeof(float)*2*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(wa),h_wa , sizeof(float)*MXMED*MXEL);
	hipMemcpyToSymbol(HIP_SYMBOL(pz), h_pz, sizeof(float)*MXMED*MXEL);
	hipMemcpyToSymbol(HIP_SYMBOL(zelem), h_zelem, sizeof(float)*MXMED*MXEL);
	//hipMemcpyToSymbol(HIP_SYMBOL(rhoz), h_rhoz, sizeof(float)*MXMED*MXEL);
	//hipMemcpyToSymbol(HIP_SYMBOL(pwr2i), h_pwr2i, sizeof(float)*MXPWR2I);
	hipMemcpyToSymbol(HIP_SYMBOL(delcm),h_delcm , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(zbrang),h_zbrang , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(lzbrang), h_lzbrang, sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(nne), h_nne, sizeof(int)*MXMED);

	hipMemcpyToSymbol(HIP_SYMBOL(smaxir), h_smaxir, sizeof(float)*MXMED);

	hipMemcpyToSymbol(HIP_SYMBOL(ibrdst), &h_ibrdst, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(iprdst), &h_iprdst, sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(ibr_nist),&h_ibr_nist , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(pair_nrc),&h_pair_nrc , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(itriplet),&h_itriplet , sizeof(int));

	//hipMemcpyToSymbol(HIP_SYMBOL(asym),h_asym , sizeof(char)*MXMED*MXEL*2);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_fdata),h_nb_fdata , sizeof(float)*MXBRXS_PLUS_1*MXBRES*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_xdata),h_nb_xdata , sizeof(float)*MXBRXS_PLUS_1*MXBRES*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_wdata),h_nb_wdata , sizeof(float)*MXBRXS*MXBRES*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_idata),h_nb_idata , sizeof(int)*MXBRXS*MXBRES*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_emin),h_nb_emin , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_emax),h_nb_emax , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_lemin),h_nb_lemin , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_lemax),h_nb_lemax , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_dle),h_nb_dle , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nb_dlei),h_nb_dlei , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(log_ap),h_log_ap , sizeof(float)*MXMED);

	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_fdata),h_nrcp_fdata , sizeof(float)*NRC_PAIR_NXX*NRC_PAIR_NEE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_wdata),h_nrcp_wdata , sizeof(float)*NRC_PAIR_NXX*NRC_PAIR_NEE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_idata),h_nrcp_idata , sizeof(int)*NRC_PAIR_NXX*NRC_PAIR_NEE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_xdata),h_nrcp_xdata , sizeof(float)*NRC_PAIR_NXX);
	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_emin),&h_nrcp_emin , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_emax),&h_nrcp_emax , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_dle),&h_nrcp_dle , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(nrcp_dlei),&h_nrcp_dlei , sizeof(float));

	//hipMemcpyToSymbol(HIP_SYMBOL(a_triplet),h_a_triplet , sizeof(float)*MAX_TRIPLET*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(b_triplet),h_b_triplet , sizeof(float)*MAX_TRIPLET*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(dl_triplet),&h_dl_triplet , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(dli_triplet),&h_dli_triplet , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(bli_triplet),&h_bli_triplet , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(log_4rm),&h_log_4rm , sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(iz_array),h_iz_array , sizeof(int)*MXTOTSH);
	hipMemcpyToSymbol(HIP_SYMBOL(be_array),h_be_array , sizeof(float)*MXTOTSH);
	hipMemcpyToSymbol(HIP_SYMBOL(jo_array),h_jo_array , sizeof(float)*MXTOTSH);
	hipMemcpyToSymbol(HIP_SYMBOL(erfjo_array),h_erfjo_array , sizeof(float)*MXTOTSH);
	hipMemcpyToSymbol(HIP_SYMBOL(ne_array),h_ne_array , sizeof(int)*MXTOTSH);
	hipMemcpyToSymbol(HIP_SYMBOL(shn_array),h_shn_array , sizeof(int)*MXTOTSH);
	hipMemcpyToSymbol(HIP_SYMBOL(shell_array),h_shell_array , sizeof(int)*MXMDSH*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(eno_array),h_eno_array , sizeof(float)*MXMDSH*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(eno_atbin_array),h_eno_atbin_array , sizeof(int)*MXMDSH*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(n_shell),h_n_shell , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_flag),&h_radc_flag , sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(binding_energies),h_binding_energies , sizeof(float)*MXSHELL*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(interaction_prob),h_interaction_prob , sizeof(float)*MXSHELL*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(relaxation_prob),h_relaxation_prob , sizeof(float)*MXTRANS*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(edge_energies),h_edge_energies , sizeof(float)*MXEDGE*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(edge_number),h_edge_number , sizeof(int)*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(edge_a),h_edge_a , sizeof(float)*MXEDGE*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(edge_b),h_edge_b , sizeof(float)*MXEDGE*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(edge_c),h_edge_c , sizeof(float)*MXEDGE*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(edge_d),h_edge_d , sizeof(float)*MXEDGE*MXELEMENT);

	//hipMemcpyToSymbol(HIP_SYMBOL(esig_e),h_esig_e , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(psig_e),h_psig_e , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(esige_max),&h_esige_max , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(psige_max),&h_psige_max , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(range_ep),h_range_ep , sizeof(float)*2*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(e_array),h_e_array , sizeof(float)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(etae_ms),h_etae_ms , sizeof(float2)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(etap_ms),h_etap_ms , sizeof(float2)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(q1ce_ms),h_q1ce_ms , sizeof(float2)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(q1cp_ms),h_q1cp_ms , sizeof(float2)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(q2ce_ms),h_q2ce_ms , sizeof(float2)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(q2cp_ms),h_q2cp_ms , sizeof(float2)*MXEKE*MXMED);

	hipMemcpyToSymbol(HIP_SYMBOL(sig_e),h_sig_e , sizeof(float)*MXMED*2);
	hipMemcpyToSymbol(HIP_SYMBOL(eta_ms),h_eta_ms , sizeof(float2)*MXEKE*MXMED*2);
	hipMemcpyToSymbol(HIP_SYMBOL(q1c_ms),h_q1c_ms , sizeof(float2)*MXEKE*MXMED*2);
	hipMemcpyToSymbol(HIP_SYMBOL(q2c_ms),h_q2c_ms , sizeof(float2)*MXEKE*MXMED*2);

	hipMemcpyToSymbol(HIP_SYMBOL(blcce),h_blcce , sizeof(float2)*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(eke01),h_eke01 , sizeof(float2)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(xr0),h_xr0 , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(teff0),h_teff0 , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(blcc),h_blcc , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(xcc),h_xcc , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(esig),h_esig , sizeof(float2)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(psig),h_psig , sizeof(float2)*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(sig),h_sig , sizeof(float2)*MXEKE*MXMED*2);
	//hipMemcpyToSymbol(HIP_SYMBOL(ededx),h_ededx , sizeof(float2)*MXEKE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(pdedx),h_pdedx , sizeof(float2)*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(dedx),h_dedx , sizeof(float2)*MXEKE*MXMED*2);
	hipMemcpyToSymbol(HIP_SYMBOL(ebr1),h_ebr1 , sizeof(float2)*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(pbr1),h_pbr1 , sizeof(float2)*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(pbr2),h_pbr2 , sizeof(float2)*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(tmxs),h_tmxs , sizeof(float2)*MXEKE*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(expeke1),h_expeke1 , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(iunrst),h_iunrst , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(epstfl),h_epstfl , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(iaprim),h_iaprim , sizeof(int)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(sig_ismonotone),h_sig_ismonotone , sizeof(bool)*2*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_xsection),h_eii_xsection , sizeof(float2)*MAX_EII_BINS);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_cons),h_eii_cons , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(eii),h_eii , sizeof(float2)*MAX_EII_SHELLS);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_z),h_eii_z , sizeof(int)*MAX_EII_SHELLS);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_sh),h_eii_sh , sizeof(int)*MAX_EII_SHELLS);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_nshells),h_eii_nshells , sizeof(int)*MXELEMENT);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_nsh),h_eii_nsh , sizeof(int)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_first),h_eii_first , sizeof(int)*MXMED*MXEL);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_no),h_eii_no , sizeof(int)*MXMED*MXEL);
	hipMemcpyToSymbol(HIP_SYMBOL(eii_flag),&h_eii_flag , sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(u_relax),&h_u_relax , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(ish_relax),&h_ish_relax , sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(iz_relax),&h_iz_relax , sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(transport_algorithm),&h_transport_algorithm , sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(bca_algorithm),&h_bca_algorithm , sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(exact_bca),&h_exact_bca , sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(spin_effects),&h_spin_effects , sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(estepe),&h_estepe , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(ximax),&h_ximax , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(skindepth_for_bca),&h_skindepth_for_bca , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(ums_array),h_ums_array , sizeof(float)*MAXL_MS_PLUS_1*MAXQ_MS_PLUS_1*MAXU_MS_PLUS_1);
	hipMemcpyToSymbol(HIP_SYMBOL(fms_array),h_fms_array , sizeof(float)*MAXL_MS_PLUS_1*MAXQ_MS_PLUS_1*MAXU_MS_PLUS_1);
	hipMemcpyToSymbol(HIP_SYMBOL(wms_array),h_wms_array , sizeof(float)*MAXL_MS_PLUS_1*MAXQ_MS_PLUS_1*MAXU_MS_PLUS_1);
	hipMemcpyToSymbol(HIP_SYMBOL(ims_array),h_ims_array , sizeof(short)*MAXL_MS_PLUS_1*MAXQ_MS_PLUS_1*MAXU_MS_PLUS_1);

	hipMemcpyToSymbol(HIP_SYMBOL(spin_rej),h_spin_rej , sizeof(float)*MXMED*2*MAXE_SPI1_PLUS_1*MAXQ_SPIN_PLUS_1*MAXU_SPIN_PLUS_1);


	hipMemcpyToSymbol(HIP_SYMBOL(llammin),&h_llammin , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(llammax),&h_llammax , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dllamb),&h_dllamb , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dllambi),&h_dllambi , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dqms),&h_dqms , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dqmsi),&h_dqmsi , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(espin_min),&h_espin_min , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(espin_max),&h_espin_max , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(espml),&h_espml , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(b2spin_min),&h_b2spin_min , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(b2spin_max),&h_b2spin_max , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dbeta2),&h_dbeta2 , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dbeta2i),&h_dbeta2i , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dlener),&h_dlener , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dleneri),&h_dleneri , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dqq1),&h_dqq1 , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dqq1i),&h_dqq1i , sizeof(float));

	//hipMemcpyToSymbol(HIP_SYMBOL(count_pii_steps),&h_count_pii_steps , sizeof(long));
	//hipMemcpyToSymbol(HIP_SYMBOL(count_all_steps),&h_count_all_steps , sizeof(long));
	//hipMemcpyToSymbol(HIP_SYMBOL(is_ch_step),&h_is_ch_step , sizeof(bool));
	//hipMemcpyToSymbol(HIP_SYMBOL(edep),&h_edep , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(tstep),&h_tstep , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(tustep),&h_tustep , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(ustep),&h_ustep , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(tvstep),&h_tvstep , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(vstep),&h_vstep , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(rhof),&h_rhof , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(eold),&h_eold , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(enew),&h_enew , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(eke01),&h_eke01 , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(elke),&h_elke , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(gle),&h_gle , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(e_range),&h_e_range , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(x_final),&h_x_final , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(y_final),&h_y_final , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(z_final),&h_z_final , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(u_final),&h_u_final , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(v_final),&h_v_final , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(w_final),&h_w_final , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(idisc),&h_idisc , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(irold),&h_irold , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(irnew),&h_irnew , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(iausfl),h_iausfl , sizeof(int)*MXAUS);

	//hipMemcpyToSymbol(HIP_SYMBOL(rlc),h_rlc , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(rldu),h_rldu , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(rho),h_rho , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(msge),h_msge , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(mge),h_mge , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(mseke),h_mseke , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(meke),h_meke , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(mleke),h_mleke , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(mcmfp),h_mcmfp , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(mrange),h_mrange , sizeof(int)*MXMED);

	//hipMemcpyToSymbol(HIP_SYMBOL(iraylm),h_iraylm , sizeof(int)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(media),h_media , sizeof(char)*24*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(photon_xsections),h_photon_xsections , sizeof(char)*16);
	hipMemcpyToSymbol(HIP_SYMBOL(apx),&h_apx , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(upx),&h_upx , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(nmed),&h_nmed , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(dunit),&h_dunit , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(kmpi),h_kmpi , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(kmpo),h_kmpo , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(ebinda),h_ebinda , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(ge),h_ge , sizeof(float2)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(gbr1),h_gbr1 , sizeof(float2)*MXGE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(gmfp),h_gmfp , sizeof(float2)*MXGE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(gbr2),h_gbr2 , sizeof(float2)*MXGE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(rco),h_rco , sizeof(float2)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(rsct),h_rsct , sizeof(float2)*MXRAYFF*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(cohe),h_cohe , sizeof(float2)*MXGE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(dpmfp),&h_dpmfp , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(mpgem),h_mpgem , sizeof(int)*MXSGE*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(ngr),h_ngr , sizeof(int)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(rmt2),&h_rmt2 , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rmsq),&h_rmsq , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(ap),h_ap , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(ae),h_ae , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(up),h_up , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(ue),h_ue , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(te),h_te , sizeof(float)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(thmoll),h_thmoll , sizeof(float)*MXMED);
	//hipMemcpyToSymbol(HIP_SYMBOL(pzero),&h_pzero , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(prm),&h_prm , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(prmt2),&h_prmt2 , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rm),&h_rm , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(medium),&h_medium , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(medold),&h_medold , sizeof(int));

	//hipMemcpyToSymbol(HIP_SYMBOL(radc_emin),&h_radc_emin , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_emax),&h_radc_emax , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_dw),&h_radc_dw , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_dle),&h_radc_dle , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_dlei),&h_radc_dlei , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_le1),&h_radc_le1 , sizeof(float));
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_sigs),&h_radc_sigs , sizeof(float)*RADC_NE_PLUS_1);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_sigd),h_radc_sigd , sizeof(float)*RADC_NE_PLUS_1);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_frej),h_radc_frej , sizeof(float)*RADC_NE_PLUS_1*RADC_NU_PLUS_1);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_x),h_radc_x , sizeof(float)*RADC_NX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_fdat),h_radc_fdat , sizeof(float)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_smax),h_radc_smax , sizeof(float)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_bins),h_radc_bins , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmin1),h_radc_ixmin1 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmax1),h_radc_ixmax1 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmin2),h_radc_ixmin2 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmax2),h_radc_ixmax2 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmin3),h_radc_ixmin3 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmax3),h_radc_ixmax3 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmin4),h_radc_ixmin4 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_ixmax4),h_radc_ixmax4 , sizeof(short)*RADC_NBOX);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_startx),h_radc_startx , sizeof(short)*RADC_NE_PLUS_1);
	//hipMemcpyToSymbol(HIP_SYMBOL(radc_startb),h_radc_startb , sizeof(short)*RADC_NE_PLUS_1);

	hipMemcpyToSymbol(HIP_SYMBOL(i_do_rr), &h_i_do_rr , sizeof(short)*MXMED);
	hipMemcpyToSymbol(HIP_SYMBOL(e_max_rr), &h_e_max_rr , sizeof(float)*MXMED);


	//hipMemcpyToSymbol(HIP_SYMBOL(prob_rr),&h_prob_rr , sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(nbr_split),&h_nbr_split , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(i_survived_rr),&h_i_survived_rr , sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(n_rr_warning),&h_n_rr_warning , sizeof(int));

	return 1;
}

