#include "hip/hip_runtime.h"


__global__ void setup_kernel(hiprandState *state)
{
    //unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
    //unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
   
	//int ThreadIndex =ix + iy *(gridDim.x*blockDim.x);

	int BlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
    int ThreadIndex =BlockIndex * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

	
	/* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(0, ThreadIndex , 0, &state[ThreadIndex ]);
	
}

void launch_setup_kernel ()
{
	d_devStates = (hiprandState**)malloc(GPUNo*sizeof(hiprandState*));

	for(int GPUId=0; GPUId<GPUNo; GPUId++) {
		
#ifdef USE_MULTIPLE_GPU
		hipSetDevice(GPUId); ce(58002);
#endif

		//int size = sizeof(hiprandState);
		//printf("  size of hiprandState . . . . %d\n  total number of thread  . . %d\n" , size, SIMULATION_NUM_THREADS);
		hipMalloc((void **)&d_devStates[GPUId], SIMULATION_NUM_THREADS*sizeof(hiprandState));

		// the grid and block configuration should match that of the simulation kernel.
		setup_kernel<<<dim3(SIMULATION_BLOCKS_PER_MULTIPROC, NUM_MULTIPROC), SIMULATION_WARPS_PER_BLOCK * WARP_SIZE>>>(d_devStates[GPUId]); ce(11000);

		hipMemcpyToSymbol(HIP_SYMBOL(devStates), &d_devStates[GPUId], sizeof(hiprandState*)); ce(9106);

	}
}

/*
void launch_setup_kernel (hiprandState * state)
{
    // execute the kernel
    //dim3 block(16, 8, 1);
    //dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //setup_kernel<<< grid, block>>>(state);

	// the grid and block configuration should match that of the simulation kernel.
	setup_kernel<<<dim3(SIMULATION_BLOCKS_PER_MULTIPROC, NUM_MULTIPROC), SIMULATION_WARPS_PER_BLOCK * WARP_SIZE>>>(state); ce(11000);

}
*/